
#include <hip/hip_runtime.h>
template <typename T> void gpuOutput(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw, int nce, int npe, int ne)
{
}

template void gpuOutput(double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int, int, int, int);
template void gpuOutput(float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int, int, int, int);

template <typename T> void __device__ devicegpuOutput(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw, int nce, int npe, int ne)
{
}

