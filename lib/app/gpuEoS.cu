
#include <hip/hip_runtime.h>
template <typename T> void gpuEoS(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw, int nce, int npe, int ne)
{
}

template void gpuEoS(double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int, int, int, int);
template void gpuEoS(float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int, int, int, int);

template <typename T> void __device__ devicegpuEoS(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw, int nce, int npe, int ne)
{
}

