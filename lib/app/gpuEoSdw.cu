
#include <hip/hip_runtime.h>
template <typename T> void gpuEoSdw(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw, int nce, int npe, int ne)
{
}

template void gpuEoSdw(double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int, int, int, int);
template void gpuEoSdw(float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int, int, int, int);

template <typename T> void __device__ devicegpuEoSdw(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw, int nce, int npe, int ne)
{
}

