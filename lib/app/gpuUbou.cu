
#include <hip/hip_runtime.h>
template <typename T>  __device__  void devicegpuUbou1(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		T param2 = param[1];
		f[0*ng+i] = param2;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T>  __global__  void kernelgpuUbou1(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	devicegpuUbou1(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template <typename T> void gpuUbou(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ib, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	if (ib == 1)
		kernelgpuUbou1<<<gridDim, blockDim>>>(f, xdg, udg, odg, wdg, uhg, nlg, tau, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template void gpuUbou(double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int, int);
template void gpuUbou(float *, float *, float *, float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int, int);
#ifdef _ENZYME
template <typename T> __global__ void kernelGradgpuUbou1Enzyme(T *f, T *df, T *xg, T *udg, T *dudg, T *odg, T *wdg, T *dwdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ib, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	__enzyme_fwddiffgpuUbou((void*)devicegpuUbou1<T>,
			  enzyme_dup, f, df,
			 enzyme_const, xg,
			 enzyme_dup, udg, dudg,
			 enzyme_const, odg,
			 enzyme_dup, wdg, dwdg,
			 enzyme_const, uhg,
			 enzyme_const, nlg,
			 enzyme_const, tau,
			 enzyme_const, uinf,
			 enzyme_const, param,
			 enzyme_const, time,
			 enzyme_const, modelnumber,
			 enzyme_const, ng,
			 enzyme_const, nc,
			 enzyme_const, ncu,
			 enzyme_const, nd,
			 enzyme_const, ncx,
			 enzyme_const, nco,
			 enzyme_const, ncw);
}

template <typename T> void gpuUbouEnzyme(T *f, T *df, T *xg, T *udg, T *dudg, T *odg, T *wdg, T *dwdg, T *uhg, T *nlg, T *tau, T *uinf, T *param, T time, int modelnumber, int ib, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	if (ib == 1)
		kernelGradgpuUbou1Enzyme<<<gridDim, blockDim>>>(f, df, xg, udg, dudg, odg, wdg, dwdg, uhg, nlg, tau, uinf, param, time, modelnumber, ib, ng, nc, ncu, nd, ncx, nco, ncw);
}

template void gpuUbouEnzyme(double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int, int);
#endif