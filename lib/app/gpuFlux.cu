
#include <hip/hip_runtime.h>
template <typename T>  __device__  void devicegpuFlux(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		T param1 = param[0];
		T udg2 = udg[1*ng+i];
		T udg3 = udg[2*ng+i];
		f[0*ng+i] = param1*udg2;
		f[1*ng+i] = param1*udg3;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> __global__ void kernelgpuFlux(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	devicegpuFlux(f, xdg, udg, odg, wdg, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template <typename T> void gpuFlux(T *f, T *xdg, T *udg, T *odg, T *wdg, T *uinf, T *param, T time, int modelnumber, int ng, int nc, int ncu, int nd, int ncx, int nco, int ncw)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuFlux<<<gridDim, blockDim>>>(f, xdg, udg, odg, wdg, uinf, param, time, modelnumber, ng, nc, ncu, nd, ncx, nco, ncw);
}

template void gpuFlux(double *, double *, double *, double *, double *, double *, double *, double, int, int, int, int, int, int, int, int);
template void gpuFlux(float *, float *, float *, float *, float *, float *, float *, float, int, int, int, int, int, int, int, int);
